#include "hip/hip_runtime.h"
/*
 * The original code is under the following copyright:
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE_GS.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 * 
 * The modifications of the code are under the following copyright:
 * Copyright (C) 2024, University of Liege, KAUST and University of Oxford
 * TELIM research group, http://www.telecom.ulg.ac.be/
 * IVUL research group, https://ivul.kaust.edu.sa/
 * VGG research group, https://www.robots.ox.ac.uk/~vgg/
 * All rights reserved.
 * The modifications are under the LICENSE.md file.
 *
 * For inquiries contact jan.held@uliege.be
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Convexes that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Convex / tile overlaps. 
// Run once per Convex (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* Convex_keys_unsorted,
	uint32_t* Convex_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Convexes
	if (radii[idx] > 0)
	{
		// Find this Convex's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Convex. Sorting the values 
		// with this key yields Convex IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				Convex_keys_unsorted[off] = key;
				Convex_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Convex ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Convexes as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P, size_t total_nb_points)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	obtain(chunk, geom.p_image, total_nb_points, 128);
	obtain(chunk, geom.hull, total_nb_points * 2, 128);
	obtain(chunk, geom.indices, total_nb_points, 128);
	obtain(chunk, geom.num_points_per_convex_view, P, 128);
	obtain(chunk, geom.offsets, total_nb_points, 128);
	obtain(chunk, geom.normals, total_nb_points, 128);
	obtain(chunk, geom.p_hom, total_nb_points, 128);
	obtain(chunk, geom.p_w, total_nb_points, 128);
	obtain(chunk, geom.p_proj, total_nb_points, 128);

	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N * 3, 128);
	obtain(chunk, img.n_contrib, N * 2, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Convexes.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* convex_points,
	const float* delta,
	const float* sigma,
	const int* num_points_per_convex,
	const int* cumsum_of_points_per_convex,
	const int total_nb_points,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	float* scaling,
	float* density_factor,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* out_others,
	int* radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P, total_nb_points);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P, total_nb_points);

	
	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Convex colors!");
	}

	// Run preprocessing per-Convex (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		convex_points,
		delta,
		sigma,
		num_points_per_convex,
		cumsum_of_points_per_convex,
		opacities,
		scaling,
		density_factor,
		shs,
		geomState.clamped,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.normals,
		geomState.offsets,
		geomState.num_points_per_convex_view,
		geomState.p_hom,
		geomState.p_w,
		geomState.p_proj,
		geomState.p_image,
		geomState.hull,
		geomState.indices,
		geomState.means2D,
		geomState.depths,
		geomState.rgb,
		geomState.conic_opacity,
		geomState.cov3D,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)

	// Compute prefix sum over full list of touched tile counts by Convexes
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Convex instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Convex indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Convex indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Convexes independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.normals,
		geomState.offsets,
		geomState.num_points_per_convex_view,
		geomState.means2D,
		delta,
		sigma,
		num_points_per_convex,
		cumsum_of_points_per_convex,
		feature_ptr,
		geomState.conic_opacity,
		geomState.depths,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		out_others), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* convex_points,
	const float* delta,
	const float* sigma,
	const int* num_points_per_convex,
	const int* cumsum_of_points_per_convex,
	const int total_nb_points,
	const float* shs,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmeans3D,
	float* dL_dmeans2D,
	float* dL_dcov3D,
	float* dL_dconvex,
	float* dL_ddelta,
	float* dL_dsigma,
	float* dL_dnormals,
	float* dL_doffsets,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dsh,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P, total_nb_points);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Convexes from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		delta, 
		sigma,
		num_points_per_convex,
		cumsum_of_points_per_convex,
		geomState.normals,
		geomState.offsets,
		geomState.num_points_per_convex_view,
		geomState.conic_opacity,
		geomState.depths,
		geomState.means2D,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float2*)dL_dnormals,
		dL_doffsets,
		dL_ddelta,
		dL_dsigma,
		(float3*)dL_dmeans2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		convex_points,
		width, height,
		radii,
		shs,
		geomState.clamped,
		viewmatrix,
		projmatrix,
		num_points_per_convex,
		cumsum_of_points_per_convex,
		geomState.p_hom,
		geomState.p_w,
		geomState.p_proj,
		geomState.p_image,
		geomState.hull,
		geomState.indices,
		geomState.num_points_per_convex_view,
		geomState.cov3D,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(glm::vec3*)dL_dconvex,
		(float2*)dL_dnormals,
		dL_doffsets,
		(glm::vec3*)dL_dmeans3D,
		(float3*)dL_dmeans2D,
		dL_dconic,
		dL_dcov3D,
		dL_dcolor,
		dL_dsh
		), debug)
}